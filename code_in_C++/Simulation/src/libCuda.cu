#include "hip/hip_runtime.h"
#include "Simulation.h"
#include "SimTypes.h"


// from https://rosettacode.org/wiki/Sutherland-Hodgman_polygon_clipping#C
#define nSteps 5 //precisão do poligono / número de pontos no poligono
#define NENV 4
#define NESPECIES 70000
#define NCELLS 20000
#define CONST_WARPS 6 //Numero arbitrario do multiplo das warps  ( corresponde a quantidade de warps em cada bloco)


#define erfA 0.278393f      // Abramowitz e Stegun approximation to erf
#define erfB 0.230389f      // https://en.wikipedia.org/wiki/Error_function
#define erfC 0.000972f
#define erfD 0.078108f
#define pi M_PI

typedef struct { 
    float x, y; 
} vec_t, *vec;

typedef struct { 
    vec v; 
    int len;
}poly_t, *poly;

__device__ float area(poly resPoly){
    float area = 0.0f;
    float d;
    int j, i;

    if (resPoly->len > 2) {
        j = resPoly->len - 1;
        for (i = 0; (i < resPoly->len); i++) {
            d = (resPoly->v[j].x + resPoly->v[i].x);
            area = area + d * (resPoly->v[j].y - resPoly->v[i].y);
            j = i;
        }
        area = -area * 0.5f;
    }
    return area;
}

__device__ void NicheCurve(float MinTol, float MidTol, float MaxTol,float MinEnv, float MaxEnv, poly NichePoly){
    // Begin of variable declarations

    float erfX;
    float erfY;
    float PhiNum;
    float PhiDen1;
    float PhiDen2;
  
    // Read input data
    float mi =    MidTol;
    float sigma = (MaxTol - mi) / 2.0;
    float a =     MinTol;
    float b =     MaxTol;
  
    float x;// = MaxTol;
    float MinimumMax = MaxTol < MaxEnv? MaxTol:MaxEnv;
    float MaximumMin = MinTol > MinEnv? MinTol:MinEnv;
    float p;
    float Tmp;
  
    float Step;
    
    // Begin of procedure
    // resPol must be nSteps+3 long
    //Step = ((b-a) / nSteps);
    Step = ((MinimumMax-MaximumMin) / nSteps);
    x = MinimumMax;
  
    NichePoly->v[0].x = x;
    NichePoly->v[0].y = 0.0f;
  
    //printf("MaximumMin=%f MinimumMax=%f\t Step=%f\n",MaximumMin,MinimumMax,Step );
    //printf("vertice %d -> x-%f   y-%f\n",0,NichePoly->v[0].x, NichePoly->v[0].y );
  
    for(int i = 0; i <= nSteps; i++){
        // https://en.wikipedia.org/wiki/Truncated_normal_distribution
        Tmp = (x - mi) / sigma;
        //PhiNum = (1.0f/sqrtf(2.0f*pi))*__expf((-0.5f)*(Tmp*Tmp));
        PhiNum = (__frsqrt_rn(2.0f*pi))*__expf((-0.5f)*(Tmp*Tmp));

        // Error function of (x1)
        erfX = ((b-mi) / sigma) / sqrtf(2.0f);
        Tmp = fabsf(erfX);

        //aqui escolher entre qual dos dois usar
        erfY = 1.0f-(1.0f/__powf(1.0f+(erfA*Tmp)+(erfB*(Tmp*Tmp))+(erfC*__powf(Tmp,3.0f))+(erfD*__powf(Tmp,4.0f)),4.0f));
        //erfY=erff(Tmp); erfY=0.95487386

        if(erfX < 0.0f){
            erfY = -1.0f * erfY;
        }

        PhiDen1 = (1.0f+erfY) / 2.0f;

        // Error function of (x2)
        erfX = ((a-mi) / sigma) / sqrt(2.0f);
        Tmp = fabs(erfX);

        //aqui escolher entre qual dos dois usar
        erfY = 1.0f-(1.0f/__powf(1.0f+(erfA*Tmp)+(erfB*(Tmp*Tmp))+(erfC*__powf(Tmp,3.0f))+(erfD*__powf(Tmp,4.0f)),4.0f));
        //erfY=erff(Tmp);

        if(erfX < 0.0f){
            erfY = -1.0f * erfY;
        }

        PhiDen2 = (1.0f+erfY) / 2.0f;

        p = (PhiNum / (sigma * (PhiDen1 - PhiDen2)));

        NichePoly->v[i+1].x = x;
        NichePoly->v[i+1].y = p;

        //printf("vertice %d -> x-%f   y-%f\n",i+1,NichePoly->v[i+1].x, NichePoly->v[i+1].y );
        x = x - Step;
    }

    NichePoly->v[nSteps+2].x = NichePoly->v[nSteps+1].x;
    NichePoly->v[nSteps+2].y = 0.0f;

    //printf("vertice %d -> x-%f   y-%f\n\n",nSteps+2,NichePoly->v[nSteps+2].x, NichePoly->v[nSteps+2].y );
}

/*__host__ __device__*/__global__ void calcFitness(float * SpNiche,float * LocEnv,float * Fitness){
    // Declare auxiliary private data

    unsigned int espIndex =(blockDim.x * blockIdx.x) + threadIdx.x; //indice da especie
    register unsigned int cellIdx;
    //sai da função se a thread não corresponde a nenhuma espécie (indice da thread maior que quantidade de especies)
    if(espIndex >= NESPECIES) return;

    float StdAreaNoOverlap, StdSimBetweenCenters;
    float MidTol;
    float MinTempTol, MaxTempTol, MinPrecpTol, MaxPrecpTol;
    float MinTempEnv, MaxTempEnv, MinPrecpEnv, MaxPrecpEnv;
    float MidEnv;
    float LocFitness;  
    // Declare private data
    vec_t NichePtns[nSteps+3];  //pontos do poligono do nicho ( da especie ) ( struct com float x e y)
    poly_t ClipedNichePoly = {NichePtns, nSteps+3};
  

    // dados da especie especifica da thread
    //armazenados assim pois são acessados varias vezes porém nunca alterados.
    MinTempTol = SpNiche[espIndex*4 +0];
    MaxTempTol = SpNiche[espIndex*4 + 1];
    MinPrecpTol = SpNiche[espIndex*4 + 2];
    MaxPrecpTol = SpNiche[espIndex*4 + 3];
          
    for(cellIdx=0;cellIdx < NCELLS ; cellIdx++){

        MinTempEnv = LocEnv[(cellIdx*4) + 0];
        MaxTempEnv = LocEnv[(cellIdx*4) + 1];
        MinPrecpEnv = LocEnv[(cellIdx*4) + 2];
        MaxPrecpEnv = LocEnv[(cellIdx*4) + 3];

        // Does the species tolerate the local environment?
        if((MinTempEnv < MinTempTol) || (MaxTempEnv > MaxTempTol)   ||   (MinPrecpEnv < MinPrecpTol) || (MaxPrecpEnv > MaxPrecpTol) ){
            LocFitness = 0.0f;
        }
        // Yes, it tolerates, lets calculate the fitness
        else {
        //Primeira variavel de ambiente
            MidTol = ((MaxTempTol + MinTempTol) / 2.0f);
            if( (MaxTempTol - MinTempTol) < 1E-2 ){
                StdAreaNoOverlap = 1;
                MidEnv = 0.0f;
            }else{
                MidEnv = ((MaxTempEnv + MinTempEnv) / 2.0f);

                //cria poligono da Tolerancia já clipado aqui
                NicheCurve(MinTempTol,MidTol,MaxTempTol,MinTempEnv,MaxTempEnv,&ClipedNichePoly);
                StdAreaNoOverlap = 1-area(&ClipedNichePoly);
            }
            StdSimBetweenCenters = 1 - (fabsf(MidEnv - MidTol) / ((MaxTempTol - MinTempTol)/2.0f));
            // Local fitness, given the first environmental variable
            LocFitness = (StdAreaNoOverlap * StdSimBetweenCenters);


        //Segunda variavel de ambiente
            MidTol = ((MaxPrecpTol + MinPrecpTol) / 2.0f);
            if( (MaxPrecpTol - MinPrecpTol) < 1E-2 ){
                StdAreaNoOverlap = 1;
                MidEnv = 0.0f;
            }else{
                MidEnv = ((MaxPrecpEnv + MinPrecpEnv) / 2.0f);

                //cria poligono da Tolerancia já clipado aqui
                NicheCurve(MinPrecpTol,MidTol,MaxPrecpTol,MinPrecpEnv,MaxPrecpEnv,&ClipedNichePoly);            
                StdAreaNoOverlap = 1 - area(&ClipedNichePoly);
            }
            StdSimBetweenCenters = 1 - (fabsf(MidEnv - MidTol) / ((MaxPrecpTol - MinPrecpTol)/2.0f));
            // Local fitness, given both environmental variables
            LocFitness = LocFitness * (StdAreaNoOverlap * StdSimBetweenCenters);
        }

        // store fitness value of 'espIndex' especie  for 'cellIdx' cell
        Fitness[ (espIndex*NCELLS) + cellIdx ] = LocFitness;
        //printf("LocFit-%.8f  CELL- %d\t Especie- %d\t storing in index %4.4i\n",LocFitness,cellIdx,espIndex, espIndex*NCELLS + cellIdx );
    }
}